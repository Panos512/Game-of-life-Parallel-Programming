#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"


#include <climits>

void printBoard(unsigned char* buffer, int width, int height)
{
	printf("----------------\n");
	for (int i = 0; i < height; i++){
		for (int j = 0; j < width; j++){
			printf("%c ", buffer[i * height + j]? 'o' : ' ');
		}
		printf("\n");
	}
	printf("----------------\n");


}

bool equal(unsigned char *array1, unsigned char *array2, int width, int height){
		for (int i = 0; i < height; i++){
			for (int j = 0; j < width; j++){
				if(array1[i * height + j] != array2[i * height + j]){
					return false;
				}
			}
		}
		printf("Evolution stoped!\n");
		return true;
}

bool empty(unsigned char *array, int width, int height){
	for (int i = 0; i < height; i++){
		for (int j = 0; j < width; j++){
			if(array[i * height + j] == 0x1){
				return false;
			}
		}
	}
	printf("Everybody died!\n");
	return true;
}

// bool equal(unsigned char *array1, unsigned char *array2, int w, int h){
// 	printf("Equal started!\n");
//     for(int i = 0; i < w * h; i++)
//         if(array1[i] != array2[i])
//             return false;
//
// 		printf("Evolution stoped!\n");
//     return true;
// }
//
// bool empty(unsigned char *array, int w, int h){
// 	printf("empty started!\n");
//     for(int i = 0 ; i < w * h; i++)
//         if(array[i] == 0x1)
//             return false;
// 		printf("Everybody died!\n");
//     return true;
// }



__global__ void golGpu(int height, int width, unsigned char* pBuffer1, unsigned char* pBuffer2){
		int x = blockIdx.x * 2 + threadIdx.x;
		int y = blockIdx.y * 2 + threadIdx.y;

		int indx = x * height + y;

		pBuffer2[indx] = pBuffer1[indx];

		int num = 0;

		if (x-1 >= 0 && x-1 < height && y >= 0 && y < width)
			num += pBuffer1[(x-1) * height + y];

		if (x+1 >= 0 && x+1 < height && y >= 0 && y < width)
			num += pBuffer1[(x+1) * height + y];

		if (x >= 0 && x < height && y-1 >= 0 && y-1 < width)
			num += pBuffer1[x * height + (y-1)];

		if (x >= 0 && x < height && y+1 >= 0 && y+1 < width)
			num += pBuffer1[x * height + (y+1)];

		if (x-1 >= 0 && x-1 < height && y-1 >= 0 && y-1 < width)
			num += pBuffer1[(x-1) * height + (y-1)];

		if (x-1 >= 0 && x-1 < height && y+1 >= 0 && y+1 < width)
			num += pBuffer1[(x-1) * height + (y+1)];

		if (x+1 >= 0 && x+1 < height && y-1 >= 0 && y-1 < width)
			num += pBuffer1[(x+1) * height + (y-1)];

		if (x+1 >= 0 && x+1 < height && y+1 >= 0 && y+1 < width)
			num += pBuffer1[(x+1) * height + (y+1)];

		if(num < 2)
			pBuffer2[indx] = 0x0;

		if(num > 3)
			pBuffer2[indx] = 0x0;

		if(num == 3 && !pBuffer1[indx])
			pBuffer2[indx] = 0x1;
		//return num;

}

void create_buffer(unsigned char* buffer, int width, int height, char** argv, int argc) {
    if (argc == 2) {
        char *filename = argv[1];
        FILE *file;

        file = fopen(filename, "r");
        if (file) {
            fscanf(file, "%*d");
            fscanf(file, "%*d");
            int arraySize = width * height;

            for (int i = 0; i < arraySize; i++) {
                int num;
                fscanf(file, "%d", &num);
                buffer[i] = num;
            }
            fclose(file);

            return;
        }

        return;


    }
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            float rnd = rand() / (float) RAND_MAX;
            buffer[i * height + j] = (rnd >= 0.7f) ? 0x1 : 0x0;
        }
    }
    return;

}


int main(int argc, char **argv){

	int width, height;
	int iterations = INT_MAX;
	// Random seed
	time_t t;
	srand((unsigned) time(&t));

	// Read file of dimensions from user (if none 12x12 is the default)
	if(argc == 2){
			char* filename = argv[1];
			FILE *file;

			file = fopen(filename, "r");
			if (file) {
					fscanf(file, "%d", &width);
					fscanf(file, "%d", &height);
			}
			fclose(file);
	}else if(argc == 3){
			width = atoi(argv[1]);
			height = atoi(argv[2]);
	}else if(argc == 4){
			width = atoi(argv[1]);
			height = atoi(argv[2]);
			iterations = atoi(argv[3]);
	}
	else{
		width = 12;
		height = 12;
	}

	//Initialise Buffer
	unsigned char* buffer;
	buffer = (unsigned char *) malloc(sizeof(unsigned char) * width * height);
  create_buffer(buffer, width, height, argv, argc);


	// printf("Starting board!\n");
	// printBoard(buffer,width,height);

	// Allocate GPU boards
	unsigned char* pBuffer1;
	hipMalloc((void **)&pBuffer1, width * height * sizeof(unsigned char));
	hipMemcpy(pBuffer1, buffer, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

	unsigned char* pBuffer2;
	hipMalloc((void **)&pBuffer2, width * height * sizeof(unsigned char));
	hipMemcpy(pBuffer2, 0x0, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

	dim3 blocksize(2, 2);
	dim3 gridsize((width + blocksize.x - 1)/blocksize.x, (height + blocksize.y - 1)/blocksize.y , 1);

	unsigned char* current;
	unsigned char* next;

	unsigned char* previeousResult;
	previeousResult = (unsigned char *)malloc(width * height * sizeof(unsigned char*));

	int gen = 0;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

	do{
		if(gen == iterations) break;
		//printf("Gen: %d\n\n\n", gen);
		memcpy(previeousResult, buffer,width * height * sizeof(unsigned char*));

		// Switching buffers to save previeous state.
		if ((gen % 2) == 0)
		{
			current = pBuffer1;
			next = pBuffer2;
		}
		else
		{
			current = pBuffer2;
			next = pBuffer1;
		}
		golGpu<<<gridsize, blocksize>>>(height, width, current, next);
		gen++;

		hipMemcpy(buffer, next, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipMemcpy(previeousResult, current, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

		//printf("Evolved\n");
		//printBoard(buffer, width, height);

		// printf("\n\nPrevieous\n\n\n");
		// printBoard(previeousResult, width, height);

	}while(!empty(buffer,width,height) && !equal(buffer, previeousResult, width, height));

	printf("Generations: %d\n", gen);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Time elapsed: %f ms\n",milliseconds);

	hipFree(pBuffer1);
	hipFree(pBuffer2);
	free(buffer);
	free(previeousResult);
	return 0;
}
